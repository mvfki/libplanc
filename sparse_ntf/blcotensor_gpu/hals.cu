#include "hals.h"

void hals_update(MAT_GPU * fm, MAT_GPU * o_mttkrp_gpu, const MAT_GPU * gram) {
  _FType *fm_times_gram_col;
  _FType *mttkrp_t; // temp for transpose mttkrp

  int m = fm->n_rows;
  int n = fm->n_cols;

  // double * _check;
  // _check = (double *) malloc(sizeof(double) * n * n);
  // hipMemcpy(_check, gram->vals, sizeof(double) * n * n, hipMemcpyDeviceToHost);
  // for (int e = 0; e < n * n; ++e) {
  //   printf("%f ", _check[e]);
  //   if ((e+1)%n == 0) printf("\n");
  // }

  check_cuda(
    hipMalloc((void**)&fm_times_gram_col, m * sizeof(_FType)), "hipMalloc fm_times_gram_col"
  );
  // transpose o_mttkrp_gpu to access rows
  check_cuda(
    hipMalloc((void**)&mttkrp_t, m * n * sizeof(_FType)), "hipMalloc mttkrp_t"
  );

  int num_elements = m * n;
  int num_blocks = (num_elements + BLOCK_SIZE - 1 )/ BLOCK_SIZE;
  __mat_transpose<<<num_blocks, BLOCK_SIZE>>>(mttkrp_t, o_mttkrp_gpu->vals, n, m);

  int rank = n;
  for (int r = 0; r < rank; ++r) {
    // get column from gram_without_one
    int col_idx = m * r;
    mat_vec_mul(fm->vals, &gram->vals[rank * r], fm_times_gram_col, m, n, 1.0, 0.0);

    num_elements = m;
    num_blocks = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dvec_sub(fm->vals+col_idx, fm_times_gram_col, fm->vals+col_idx, m);
    dvec_add(fm->vals+col_idx, mttkrp_t+col_idx, fm->vals+col_idx, m);
    __apply_threshold<<<num_blocks, BLOCK_SIZE>>>(fm->vals+col_idx, m, 1e-16, 1e-16);
  }

  hipFree(fm_times_gram_col);
  hipFree(mttkrp_t);
}