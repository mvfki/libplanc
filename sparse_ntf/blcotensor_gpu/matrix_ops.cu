#include "hip/hip_runtime.h"
#include "matrix_ops.h"

static const double one = 1.0;
static const double zero = 0.0;


// Takes the reciprocal of each vector entry, sets to zero if smaller than tol
__global__ void reciprocal_vector_kernel(double * v, unsigned int n, double tol) {
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
    v[index] = (fabs(v[index]) > tol) ? 1.0 / v[index] : 0;
  }
}

// Use hipsolverDnDgesvdj_bufferSize to calculate needed buffer size
void pseudoinverse_gpu(hipsolverHandle_t cusolverHandle, hipblasHandle_t cublasHandle,
    hipStream_t stream, double* A, unsigned int n, double* work, unsigned int lwork, int* info, hipsolverGesvdjInfo_t gesvd_info) {

    // I tried Cholesky / QR / LU factorization
    // They scale poorly to larger matrices compared to svd + gemm

    double* U = work;
    double* V = U + n * n;
    double* S = V + n * n;
    work = S + n;
    lwork -= (2 * n * n + n);

    // Gen SVD
    check_cublas(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE), "hipblasSetPointerMode");
    check_cusolver(hipsolverDnDgesvdj(cusolverHandle, HIPSOLVER_EIG_MODE_VECTOR, 
        0, n, n, A, n, S, U, n, V, n, work, lwork, info, gesvd_info), "hipsolverDnDgesvdj");

    check_cuda(hipStreamSynchronize(stream), "cusolverDngesvdj execute");

    // Multiply U by S^-1 (scale rows of U by reciprocal of S);
    unsigned int blocks = n / BLOCK_SIZE + 1;
    double s = 0; // Get largest singular value
    check_cuda(hipMemcpy(&s, S, sizeof(double), hipMemcpyDeviceToHost), "memcpy");
    s = n * (nextafter(s, s + 1) - s);
    reciprocal_vector_kernel <<<blocks, BLOCK_SIZE, 0, stream>>>(S, n, s);
    //check_cuda(hipGetLastError(), "reciprocal_vector launch");
    check_cuda(hipStreamSynchronize(stream), "reciprocal_vector execute");

    // Multiply U * S
    for (unsigned int i = 0; i < n; i++) {
        hipblasSetStream(cublasHandle, stream);
        #ifdef USE_32BIT_TYPE
            check_cublas(hipblasSscal(cublasHandle, n, S + i, U + i * n, 1), "hipblasSscal");
        #else 
            check_cublas(hipblasDscal(cublasHandle, n, S + i, U + i * n, 1), "hipblasDscal");
        #endif
    }
    check_cuda(hipStreamSynchronize(stream), "hipblasDscal execute");

    // Multiply U by V (we multiply V by U^T to convert col to row major)
    check_cublas(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST), "hipblasSetPointerMode");
    hipblasSetStream(cublasHandle, stream);

    check_cublas(hipblasDgemm(
        cublasHandle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        n, n, n,
        &one, 
        V, n,
        U, n,
        &zero, 
        A, n), "hipblasDgemm"
    );
    check_cuda(hipStreamSynchronize(stream), "hipblasDgemm execute");
}


void solveSvdGemm(const MAT_GPU &A, const MAT_GPU &B, MAT_GPU &X) {

  hipStream_t v_stream;
  hipblasHandle_t cublasHandle;
  hipsolverHandle_t cusolverHandle;
  check_cuda(hipStreamCreate(&v_stream), "hipStreamCreate");
  check_cublas(hipblasCreate(&cublasHandle), "hipblasCreate");
  check_cusolver(hipsolverDnCreate(&cusolverHandle), "hipsolverDnCreate");
  
  // Set stream
  hipblasSetStream(cublasHandle, v_stream);
  hipsolverSetStream(cusolverHandle, v_stream);

  // Allocate pseudoinverse array + work
  double *work;

  int work_int = 0;
  assert(A.n_rows == A.n_cols); // A should be square matrix

  int m = A.n_rows;
  int k = A.n_cols;
  int n = B.n_cols;

  check_cublas(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST), "hipblasSetPointerMode");
  hipsolverGesvdjInfo_t gesvd_info;

  check_cusolver(hipsolverDnCreateGesvdjInfo(&gesvd_info), "hipsolverDnCreateGesvdjInfo");
  hipsolverDnXgesvdjSetMaxSweeps(gesvd_info, 15); // As recommended by cuSOLVER docs

  check_cusolver(
    hipsolverDnDgesvdj_bufferSize(cusolverHandle, HIPSOLVER_EIG_MODE_VECTOR, 1, m, m, A.vals, m, NULL, NULL, m, NULL, m, &work_int, gesvd_info), "hipsolverDnDgesvdj_bufferSize");
  unsigned int work_length = 2 * m * m + m + work_int;
  check_cuda(hipMalloc(&work, sizeof(double) * work_length), "hipMalloc work");
  int * info;
  check_cuda(hipMalloc(&info, sizeof(int)), "hipMalloc info");
  // double * A
  // unsigned int n
  // double * work
  // unsigned int lwork
  // int * info
  // hipsolverGesvdjInfo_t

  // Take pseudoinverse
  pseudoinverse_gpu(cusolverHandle, cublasHandle, v_stream, 
    A.vals, m, work, work_length, info, gesvd_info);

  // Multiply V^-1 by b
  check_cublas(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, 
    n, k, &one, A.vals, m, B.vals, k, &zero, X.vals, m), "hipblasDgemm");

  // Clean up
  hipblasDestroy(cublasHandle);
  hipsolverDnDestroy(cusolverHandle);
  hipsolverDnDestroyGesvdjInfo(gesvd_info);
  hipStreamDestroy(v_stream);
  hipFree(work);
  hipFree(info);
}
