#include "cuda_utils.h"

template <typename T>
T* make_device_copy(T* vector, _IType n, std::string name) {
    T* d_vector = nullptr;
    check_cuda(hipMalloc(&d_vector, sizeof(T) * n), "hipMalloc " + name);
    check_cuda(hipMemcpy(d_vector, vector, sizeof(T) * n, hipMemcpyHostToDevice), "hipMemcpy " + name);
    return d_vector;
};
template _IType* make_device_copy(_IType* vector, _IType n, std::string name);
template _FType* make_device_copy(_FType* vector, _IType n, std::string name);
template _FType** make_device_copy(_FType** vector, _IType n, std::string name);
template _FType*** make_device_copy(_FType*** vector, _IType n, std::string name);
template unsigned int* make_device_copy(unsigned int* vector, _IType n, std::string name);
template unsigned long* make_device_copy(unsigned long* vector, _IType n, std::string name);
template int* make_device_copy(int* vector, _IType n, std::string name);


void check_cublas(hipblasStatus_t status, std::string message) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Error: " << cublasGetStatusString(status);
    std::cerr << ". " << message << std::endl;
    exit(EXIT_FAILURE);
  }
}

void check_cuda(hipError_t status, std::string message) {
  if (status != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(status);
    std::cerr << ". " << message << std::endl;
    exit(EXIT_FAILURE);
  }
}
